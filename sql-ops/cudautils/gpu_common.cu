#include "hip/hip_runtime.h"
/*
 * This Source Code Form is subject to the terms of the Mozilla Public License,
 * v. 2.0. If a copy of the MPL was not distributed with this file, You can
 * obtain one at http://mozilla.org/MPL/2.0/.
 *
 *
 * Copyright (c) 2020 Clemens Lutz, German Research Center for Artificial
 * Intelligence
 * Author: Clemens Lutz, DFKI GmbH <clemens.lutz@dfki.de>
 */

#include <gpu_common.h>

// Returns the log2 of the next-lower power of two
__device__ int log2_floor_power_of_two(int x) { return 32 - __clz(x) - 1; }

// Returns the log2 of the next-higher power of two
__device__ int log2_ceil_power_of_two(int x) { return 32 - __clz(x - 1); }
