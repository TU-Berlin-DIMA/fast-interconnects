#include "hip/hip_runtime.h"
/*
 * This Source Code Form is subject to the terms of the Mozilla Public License,
 * v. 2.0. If a copy of the MPL was not distributed with this file, You can
 * obtain one at http://mozilla.org/MPL/2.0/.
 *
 *
 * Copyright (c) 2020 Clemens Lutz, German Research Center for Artificial
 * Intelligence
 * Author: Clemens Lutz, DFKI GmbH <clemens.lutz@dfki.de>
 */

#define CUDA_MODIFIER __device__
// #define DEBUG

#include <gpu_common.h>
#include <gpu_radix_partition.h>

#include <cassert>
#include <climits>
#include <cstdint>

#ifdef DEBUG
#include <cstdio>
#endif

using namespace std;

// Arguments to the join-aggregate function.
//
// Note that the struct's layout must be kept in sync with its counterpart in
// Rust.
struct JoinAggregateArgs {
  void const *const build_rel;
  uint64_t const *const build_rel_partition_offsets;
  void const *const probe_rel;
  uint64_t const *const probe_rel_partition_offsets;
  int64_t *const aggregation_result;
  uint32_t *const task_assignments;
  uint32_t const build_rel_length;
  uint32_t const probe_rel_length;
  uint32_t const build_rel_padding_length;
  uint32_t const probe_rel_padding_length;
  uint32_t const radix_bits;
  uint32_t const ignore_bits;
  uint32_t const ht_entries;
};

// Assign tasks to thread blocks
//
// Each task is a contiguous sequence of partitions. This assumes that there
// are more partitions than thread blocks. If the assumption does not hold,
// then some thread blocks are assigned an empty task (i.e.,
// task_assignments[i] == task_assignments[i+1]). The last array index is a
// sentinal value that is set to the fanout.
extern "C" __global__ void gpu_radix_join_assign_tasks(JoinAggregateArgs args) {
  // FIXME: parallelize task assignment
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    const uint32_t fanout = 1U << args.radix_bits;
    const uint32_t probe_tuples =
        args.probe_rel_length - args.probe_rel_padding_length * fanout;
    const uint32_t avg_task_size = (probe_tuples + gridDim.x - 1U) / gridDim.x;

    args.task_assignments[0] = 0U;
    uint32_t task_id = 1U;
    uint32_t task_size = 0U;
    for (uint32_t p = 0U; p < fanout && task_id < gridDim.x; p += 1U) {
      uint32_t probe_upper = (p + 1 < fanout)
                                 ? args.probe_rel_partition_offsets[p + 1U] -
                                       args.probe_rel_padding_length
                                 : args.probe_rel_length;
      uint32_t probe_size = static_cast<uint32_t>(
          probe_upper - args.probe_rel_partition_offsets[p]);

      task_size += probe_size;
      if (task_size >= avg_task_size) {
        args.task_assignments[task_id] = p + 1U;

#ifdef DEBUG
        printf("Assigning partitions [%u, %u] to block %d\n",
               args.task_assignments[task_id - 1],
               args.task_assignments[task_id], task_id);
#endif

        task_size = 0U;
        task_id += 1;
      }
    }

    // assign an empty task if fanout < gridDim.x
    // and initialize sentinal value at task_assignments[gridDim.x]
    for (uint32_t tid = task_id; tid <= gridDim.x; tid += 1U) {
      args.task_assignments[tid] = fanout;
    }
  }
}

// Bucket chaining hash join in shared memory.
//
// See the Rust module for details.
template <typename K, typename PI, typename PO>
__device__ void gpu_radix_join_aggregate_smem_perfect(JoinAggregateArgs &args) {
  extern __shared__ uint32_t shared_mem[];

  const uint32_t fanout = 1U << args.radix_bits;
  const uint64_t mask = ~static_cast<uint64_t>((1U << args.ignore_bits) - 1U);

  HtEntry<K, PI> *const __restrict__ hash_table =
      reinterpret_cast<HtEntry<K, PI> *>(shared_mem);

  int64_t sum = 0;

  for (uint32_t p = args.task_assignments[blockIdx.x];
       p < args.task_assignments[blockIdx.x + 1U]; p += 1U) {
    Tuple<K, PI> const *const __restrict__ build_rel =
        reinterpret_cast<Tuple<K, PI> const *>(args.build_rel) +
        args.build_rel_partition_offsets[p];
    Tuple<K, PO> const *const __restrict__ probe_rel =
        reinterpret_cast<Tuple<K, PO> const *>(args.probe_rel) +
        args.probe_rel_partition_offsets[p];

    uint32_t build_upper = (p + 1U < fanout)
                               ? args.build_rel_partition_offsets[p + 1U] -
                                     args.build_rel_padding_length
                               : args.build_rel_length;
    uint32_t build_size = static_cast<uint32_t>(
        build_upper - args.build_rel_partition_offsets[p]);

    assert(build_size <= args.ht_entries &&
           "Build-side relation is larger than hash table");

    uint32_t probe_upper = (p + 1U < fanout)
                               ? args.probe_rel_partition_offsets[p + 1U] -
                                     args.probe_rel_padding_length
                               : args.probe_rel_length;
    uint32_t probe_size = static_cast<uint32_t>(
        probe_upper - args.probe_rel_partition_offsets[p]);

#ifdef DEBUG
    if (threadIdx.x == 0) {
      printf("part: %d, fanout: %d, build_size: %d, probe_size: %d\n", p,
             fanout, build_size, probe_size);
    }
#endif

    // Initialize hash table
    for (uint32_t i = threadIdx.x; i < args.ht_entries; i += blockDim.x) {
      hash_table[i] = {null_key<K>(), 0};
    }

    __syncthreads();

    // Build
    for (uint32_t i = threadIdx.x; i < build_size; i += blockDim.x) {
      Tuple<K, PI> tuple = build_rel[i];
      auto ht_index = key_to_partition(tuple.key, mask, args.ignore_bits);

#ifdef DEBUG
      assert(ht_index < args.ht_entries && "Invalid hash table index");
#endif

      hash_table[ht_index] = {tuple.key, tuple.value};
    }

    __syncthreads();

    // Probe
    for (uint32_t i = threadIdx.x; i < probe_size; i += blockDim.x) {
      Tuple<K, PO> tuple = probe_rel[i];
      auto ht_index = key_to_partition(tuple.key, mask, args.ignore_bits);

#ifdef DEBUG
      assert(ht_index < args.ht_entries && "Invalid hash table index");
#endif

      if (hash_table[ht_index].key == tuple.key) {
        sum += tuple.value;
      }
#ifdef DEBUG
      else {
        printf(
            "tid: %u, part: %u, ht_index: %u, build_key: %u, probe_key: %u\n",
            threadIdx.x, p, ht_index, hash_table[ht_index].key, tuple.key);
      }
#endif
    }

    __syncthreads();
  }

  args.aggregation_result[blockDim.x * blockIdx.x + threadIdx.x] += sum;
}

// Bucket chaining hash join in shared memory.
//
// See the Rust module for details.
template <typename K, typename PI, typename PO>
__device__ void gpu_radix_join_aggregate_smem_chaining(
    JoinAggregateArgs &args, const uint32_t shared_mem_bytes) {
  extern __shared__ uint32_t shared_mem[];

  const uint32_t fanout = 1U << args.radix_bits;
  const uint64_t mask = ~static_cast<uint64_t>((1U << args.ignore_bits) - 1U);
  constexpr unsigned short tail = USHRT_MAX;

  const uint32_t buckets = args.ht_entries;
  assert(buckets * sizeof(unsigned int) < shared_mem_bytes &&
         "The hash table buckets are larger than shared memory, reduce the "
         "ht_entries tuning parameter.");
  size_t ht_bytes = shared_mem_bytes - buckets * sizeof(unsigned int);
  const uint32_t ht_entries =
      ht_bytes / (sizeof(unsigned short) + sizeof(K) + sizeof(PI));
  assert(ht_entries >= 1 &&
         "Number of hash table entries is too small; try reducing the number "
         "of hash table buckets");
  const unsigned int buckets_mask = buckets - 1U;
#ifdef DEBUG
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    printf("Number of HT buckets: %u, number of entries: %u\n", buckets,
           ht_entries);
  }
#endif

  K *const __restrict__ keys = reinterpret_cast<K *>(shared_mem);
  PI *const __restrict__ values = reinterpret_cast<PI *>(&keys[ht_entries]);
  unsigned int *const __restrict__ heads =
      reinterpret_cast<unsigned int *>(&values[ht_entries]);
  unsigned short *const __restrict__ links =
      reinterpret_cast<unsigned short *>(&heads[buckets]);

  int64_t sum = 0;

  for (uint32_t p = args.task_assignments[blockIdx.x];
       p < args.task_assignments[blockIdx.x + 1U]; p += 1U) {
    Tuple<K, PI> const *const __restrict__ build_rel =
        reinterpret_cast<Tuple<K, PI> const *>(args.build_rel) +
        args.build_rel_partition_offsets[p];
    Tuple<K, PO> const *const __restrict__ probe_rel =
        reinterpret_cast<Tuple<K, PO> const *>(args.probe_rel) +
        args.probe_rel_partition_offsets[p];

    uint32_t build_upper = (p + 1U < fanout)
                               ? args.build_rel_partition_offsets[p + 1U] -
                                     args.build_rel_padding_length
                               : args.build_rel_length;
    uint32_t build_size = static_cast<uint32_t>(
        build_upper - args.build_rel_partition_offsets[p]);

    assert(build_size <= ht_entries &&
           "Build-side relation is larger than hash table");

    uint32_t probe_upper = (p + 1U < fanout)
                               ? args.probe_rel_partition_offsets[p + 1U] -
                                     args.probe_rel_padding_length
                               : args.probe_rel_length;
    uint32_t probe_size = static_cast<uint32_t>(
        probe_upper - args.probe_rel_partition_offsets[p]);

#ifdef DEBUG
    if (threadIdx.x == 0) {
      printf("part: %d, fanout: %d, build_size: %d, probe_size: %d\n", p,
             fanout, build_size, probe_size);
    }
#endif

    // Initialize hash table
    for (uint32_t i = threadIdx.x; i < buckets; i += blockDim.x) {
      heads[i] = static_cast<unsigned int>(tail);
    }

    __syncthreads();

    // Build
    for (unsigned int i = threadIdx.x; i < build_size; i += blockDim.x) {
      Tuple<K, PI> tuple;
      tuple.load(build_rel[i]);

      keys[i] = tuple.key;
      values[i] = tuple.value;

      auto ht_index = key_to_partition(tuple.key, mask, args.ignore_bits);
      auto bucket = hash<K>(ht_index) & buckets_mask;
      unsigned int next = atomicExch(&heads[bucket], i);
      links[i] = static_cast<unsigned short>(next);
    }

    __syncthreads();

    // Probe
    for (uint32_t i = threadIdx.x; i < probe_size; i += blockDim.x) {
      Tuple<K, PO> tuple;
      tuple.load(probe_rel[i]);

      auto ht_index = key_to_partition(tuple.key, mask, args.ignore_bits);
      auto bucket = hash<K>(ht_index) & buckets_mask;

      for (unsigned short i = static_cast<unsigned short>(heads[bucket]);
           i != tail; i = links[i]) {
        if (keys[i] == tuple.key) {
          sum += tuple.value;
        }
      }
    }

    __syncthreads();
  }

  args.aggregation_result[blockDim.x * blockIdx.x + threadIdx.x] += sum;
}

extern "C" __global__ void gpu_join_aggregate_smem_perfect_i32_i32_i32(
    JoinAggregateArgs args) {
  gpu_radix_join_aggregate_smem_perfect<int, int, int>(args);
}

extern "C" __global__ void gpu_join_aggregate_smem_perfect_i64_i64_i64(
    JoinAggregateArgs args) {
  gpu_radix_join_aggregate_smem_perfect<long long, long long, long long>(args);
}

extern "C" __global__ void gpu_join_aggregate_smem_chaining_i32_i32_i32(
    JoinAggregateArgs args, const uint32_t shared_mem_bytes) {
  gpu_radix_join_aggregate_smem_chaining<int, int, int>(args, shared_mem_bytes);
}

extern "C" __global__ void gpu_join_aggregate_smem_chaining_i64_i64_i64(
    JoinAggregateArgs args, const uint32_t shared_mem_bytes) {
  gpu_radix_join_aggregate_smem_chaining<long long, long long, long long>(
      args, shared_mem_bytes);
}
