/*
 * This Source Code Form is subject to the terms of the Mozilla Public License,
 * v. 2.0. If a copy of the MPL was not distributed with this file, You can
 * obtain one at http://mozilla.org/MPL/2.0/.
 *
 *
 * Copyright (c) 2020, Clemens Lutz <lutzcle@cml.li>
 * Author: Clemens Lutz <clemens.lutz@dfki.de>
 */


#include <hip/hip_runtime.h>
#include <cstdint>

extern "C" __global__ void tpch_q6_branching(
    uint64_t length, int32_t *l_shipdate, int32_t *l_discount,
    int32_t *l_quantity, int32_t *l_extendedprice, uint64_t *revenue,
    uint64_t *negative_revenue) {
  const uint32_t global_idx = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t global_threads = blockDim.x * gridDim.x;

  __shared__ unsigned long long block_revenue;
  __shared__ unsigned long long block_negative_revenue;

  if (threadIdx.x == 0) {
    block_revenue = 0;
    block_negative_revenue = 0;
  }
  __syncthreads();

  // Parallel query computation
  long long private_revenue = 0;
  for (uint64_t i = global_idx; i < length; i += global_threads) {
    if (l_shipdate[i] >= 366 + 365 + 1 && l_shipdate[i] < 366 + 365 + 365 + 1 &&
        l_discount[i] >= 5 && l_discount[i] <= 7 && l_quantity[i] < 24) {
      private_revenue += l_extendedprice[i] * l_discount[i];
    }
  }

  // Reduce result, with work-around because CUDA doesn't support atomicAdd for
  // long long int (i.e., signed 64-bit integers)
  if (private_revenue >= 0) {
    atomicAdd(&block_revenue, private_revenue);
  } else {
    atomicAdd(&block_negative_revenue, -private_revenue);
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    if (block_revenue > 0) {
      atomicAdd(reinterpret_cast<unsigned long long *>(revenue), block_revenue);
    }
    if (block_negative_revenue > 0) {
      atomicAdd(reinterpret_cast<unsigned long long *>(negative_revenue),
                block_negative_revenue);
    }
  }
}

extern "C" __global__ void tpch_q6_predication(
    uint64_t length, int32_t *l_shipdate, int32_t *l_discount,
    int32_t *l_quantity, int32_t *l_extendedprice, uint64_t *revenue,
    uint64_t *negative_revenue) {
  const uint32_t global_idx = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t global_threads = blockDim.x * gridDim.x;

  __shared__ unsigned long long block_revenue;
  __shared__ unsigned long long block_negative_revenue;

  if (threadIdx.x == 0) {
    block_revenue = 0;
    block_negative_revenue = 0;
  }
  __syncthreads();

  // Parallel query computation
  long long private_revenue = 0;
  for (uint64_t i = global_idx; i < length; i += global_threads) {
    int condition = (l_shipdate[i] >= 366 + 365 + 1) &
                    (l_shipdate[i] < 366 + 365 + 365 + 1) &
                    (l_discount[i] >= 5) & (l_discount[i] <= 7) &
                    (l_quantity[i] < 24);
    condition = ((!condition) << 31) >> 31;
    private_revenue += condition & (l_extendedprice[i] * l_discount[i]);
  }

  // Reduce result, with work-around because CUDA doesn't support atomicAdd for
  // long long int (i.e., signed 64-bit integers)
  if (private_revenue >= 0) {
    atomicAdd(&block_revenue, private_revenue);
  } else {
    atomicAdd(&block_negative_revenue, -private_revenue);
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    if (block_revenue > 0) {
      atomicAdd(reinterpret_cast<unsigned long long *>(revenue), block_revenue);
    }
    if (block_negative_revenue > 0) {
      atomicAdd(reinterpret_cast<unsigned long long *>(negative_revenue),
                block_negative_revenue);
    }
  }
}
