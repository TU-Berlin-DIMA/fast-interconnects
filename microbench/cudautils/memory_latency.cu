// Copyright 2018-2022 Clemens Lutz
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.


#include <hip/hip_runtime.h>
#include <cstdint>

// X mod Y, assuming that Y is a power of 2
#define FAST_MODULO(X, Y) (X & (Y - 1))

__global__ void gpu_stride(uint32_t *data, uint32_t iterations,
                           uint64_t *cycles) {
  uint64_t sum = 0;
  uint64_t start = 0;
  uint64_t stop = 0;
  uint32_t pos = 0;
  uint32_t dependency = 0;  // Prevent compiler from optimizing away the loop

  // Warm-up the cache
  for (uint32_t i = 0; i < iterations; ++i) {
    pos = data[pos];
    dependency += pos;
  }

  // Prevent optimization and reset position
  if (pos != 0) {
    pos = 0;
  }

  start = clock64();

  // Do measurement
  for (uint32_t i = 0; i < iterations; ++i) {
    pos = data[pos];
    dependency += pos;
  }

  stop = clock64();
  sum += stop - start;

  // Write result
  *cycles = (uint32_t)(sum / ((uint64_t)iterations));

  // Prevent compiler optimization
  if (pos == 1) {
    data[1] = dependency;
  }
}
