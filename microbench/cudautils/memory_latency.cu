
#include <hip/hip_runtime.h>
#include <cstdint>

// X mod Y, assuming that Y is a power of 2
#define FAST_MODULO(X, Y) (X & (Y - 1))

__global__ void gpu_stride(uint32_t *data, uint32_t iterations,
                           uint64_t *cycles) {
  uint64_t sum = 0;
  uint64_t start = 0;
  uint64_t stop = 0;
  uint32_t pos = 0;
  uint32_t dependency = 0;  // Prevent compiler from optimizing away the loop

  // Warm-up the cache
  for (uint32_t i = 0; i < iterations; ++i) {
    pos = data[pos];
    dependency += pos;
  }

  // Prevent optimization and reset position
  if (pos != 0) {
    pos = 0;
  }

  start = clock64();

  // Do measurement
  for (uint32_t i = 0; i < iterations; ++i) {
    pos = data[pos];
    dependency += pos;
  }

  stop = clock64();
  sum += stop - start;

  // Write result
  *cycles = (uint32_t)(sum / ((uint64_t)iterations));

  // Prevent compiler optimization
  if (pos == 1) {
    data[1] = dependency;
  }
}
